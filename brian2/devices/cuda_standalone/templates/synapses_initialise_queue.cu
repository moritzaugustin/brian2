#include "hip/hip_runtime.h"
{% macro cu_file() %}
#include "code_objects/{{codeobj_name}}.h"
{% set pathobj = owner.name %}

namespace {
	int num_blocks(int objects){
		return ceil(objects / (double)brian::max_threads_per_block);
	}
	int num_threads(int objects){
		return brian::max_threads_per_block;
	}
}

__global__ void _run_{{codeobj_name}}_kernel(
	unsigned int _target_N,
	unsigned int _num_blocks,
	unsigned int _num_threads_per_block,
	double _dt,
	unsigned int _syn_N,
	unsigned int max_delay)
{
	using namespace brian;

	int tid = threadIdx.x;

	{{pathobj}}.queue->prepare(
		tid,
		_num_threads_per_block,
		_num_blocks,
		_dt,
		_target_N,
		_syn_N,
		max_delay,
		{{pathobj}}_size_by_pre,
		{{pathobj}}_synapses_id_by_pre,
		{{pathobj}}_delay_by_pre);
}

//POS(queue_id, neuron_id, neurons_N)
#define OFFSET(a, b, c)	(a*c + b)

void _run_{{pathobj}}_initialise_queue()
{
	using namespace brian;

	double dt = {{owner._clock._name}}.dt_();
	unsigned int syn_N = dev_dynamic_array_{{pathobj}}_delay.size();
	unsigned int source_N = {{owner.source.N}};
	unsigned int target_N = {{owner.target.N}};

	//Create temporary host vectors
	int32_t* h_synapses_synaptic_sources = new int32_t[syn_N];
	int32_t* h_synapses_synaptic_targets = new int32_t[syn_N];
	double* h_synapses_delay = new double[syn_N];

	hipMemcpy(h_synapses_synaptic_sources, thrust::raw_pointer_cast(&dev_dynamic_array_{{owner.synapses.name}}_{{owner.synapse_sources.name}}[0]), sizeof(int32_t) * syn_N, hipMemcpyDeviceToHost);
	hipMemcpy(h_synapses_synaptic_targets, thrust::raw_pointer_cast(&dev_dynamic_array_{{owner.synapses.name}}_{{owner.synapse_targets.name}}[0]), sizeof(int32_t) * syn_N, hipMemcpyDeviceToHost);
	hipMemcpy(h_synapses_delay, thrust::raw_pointer_cast(&dev_dynamic_array_{{pathobj}}_delay[0]), sizeof(double) * syn_N, hipMemcpyDeviceToHost);
	{% if no_delay_mode == False%}
	thrust::host_vector<int32_t>* h_synapses_by_pre_id = new thrust::host_vector<int32_t>[num_parallel_blocks*source_N];
	thrust::host_vector<unsigned int>* h_delay_by_pre_id = new thrust::host_vector<unsigned int>[num_parallel_blocks*source_N];
	{% else %}
	num_parallel_blocks = 1;
	thrust::host_vector<int32_t>* h_synapses_by_pre_id = new thrust::host_vector<int32_t>[target_N];
	thrust::host_vector<unsigned int>* h_delay_by_pre_id = new thrust::host_vector<unsigned int>[target_N];
	{% endif %}

	//fill vectors with pre_neuron, post_neuron, delay data
	unsigned int max_delay = 0;
	for(int syn_id = 0; syn_id < syn_N; syn_id++)
	{
		int32_t pre_neuron_id = h_synapses_synaptic_sources[syn_id] - {{owner.source.start}};
		int32_t post_neuron_id = h_synapses_synaptic_targets[syn_id]  - {{owner.target.start}};
		{% if no_delay_mode == False%}
		unsigned int delay = (int)(h_synapses_delay[syn_id] / dt + 0.5);
		if(delay > max_delay)
		{
			max_delay = delay;
		}
		unsigned int right_queue = (post_neuron_id*num_parallel_blocks)/target_N;
		unsigned int right_offset = pre_neuron_id * num_parallel_blocks + right_queue;
		h_synapses_by_pre_id[right_offset].push_back(syn_id);
		h_delay_by_pre_id[right_offset].push_back(delay);
		{% else %}
		h_synapses_by_pre_id[post_neuron_id].push_back(syn_id);
		h_delay_by_pre_id[post_neuron_id].push_back(0);
		{% endif %}
	}
	max_delay++;	//we also need a current step

	//create array for device pointers
	{% if no_delay_mode == False%}
	unsigned int* temp_size_by_pre_id = new unsigned int[num_parallel_blocks*source_N];
	int32_t** temp_synapses_by_pre_id = new int32_t*[num_parallel_blocks*source_N];
	unsigned int** temp_delay_by_pre_id = new unsigned int*[num_parallel_blocks*source_N];
	//fill temp arrays with device pointers
	for(int i = 0; i < num_parallel_blocks*source_N; i++)
	{
		int num_elements = h_synapses_by_pre_id[i].size();
		temp_size_by_pre_id[i] = num_elements;
		if(num_elements > 0)
		{
			hipMalloc((void**)&temp_synapses_by_pre_id[i], sizeof(int32_t)*num_elements);
			hipMalloc((void**)&temp_delay_by_pre_id[i], sizeof(unsigned int)*num_elements);
			hipMemcpy(temp_synapses_by_pre_id[i],
				thrust::raw_pointer_cast(&(h_synapses_by_pre_id[i][0])),
				sizeof(int32_t)*num_elements,
				hipMemcpyHostToDevice);
			hipMemcpy(temp_delay_by_pre_id[i],
				thrust::raw_pointer_cast(&(h_delay_by_pre_id[i][0])),
				sizeof(unsigned int)*num_elements,
				hipMemcpyHostToDevice);
		}
	}

	//copy temp arrays to device
	unsigned int* temp;
	hipMalloc((void**)&temp, sizeof(unsigned int)*num_parallel_blocks*source_N);
	hipMemcpy(temp, temp_size_by_pre_id, sizeof(unsigned int)*num_parallel_blocks*source_N, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL({{pathobj}}_size_by_pre), &temp, sizeof(unsigned int*));
	int32_t* temp2;
	hipMalloc((void**)&temp2, sizeof(int32_t*)*num_parallel_blocks*source_N);
	hipMemcpy(temp2, temp_synapses_by_pre_id, sizeof(int32_t*)*num_parallel_blocks*source_N, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL({{pathobj}}_synapses_id_by_pre), &temp2, sizeof(int32_t**));
	unsigned int* temp3;
	hipMalloc((void**)&temp3, sizeof(unsigned int*)*num_parallel_blocks*source_N);
	hipMemcpy(temp3, temp_delay_by_pre_id, sizeof(int32_t*)*num_parallel_blocks*source_N, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL({{pathobj}}_delay_by_pre), &temp3, sizeof(unsigned int**));
	{% else %}
	//NO DELAY MODE
	unsigned int* temp_size_by_pre_id = new unsigned int[target_N];
	int32_t** temp_synapses_by_pre_id = new int32_t*[target_N];
	unsigned int** temp_delay_by_pre_id = new unsigned int*[target_N];
	//fill temp arrays with device pointers
	for(int i = 0; i < target_N; i++)
	{
		int num_elements = h_synapses_by_pre_id[i].size();
		temp_size_by_pre_id[i] = num_elements;
		if(num_elements > 0)
		{
			hipMalloc((void**)&temp_synapses_by_pre_id[i], sizeof(int32_t)*num_elements);
			hipMemcpy(temp_synapses_by_pre_id[i],
				thrust::raw_pointer_cast(&(h_synapses_by_pre_id[i][0])),
				sizeof(int32_t)*num_elements,
				hipMemcpyHostToDevice);
		}
	}

	//copy temp arrays to device
	unsigned int* temp;
	hipMalloc((void**)&temp, sizeof(unsigned int)*target_N);
	hipMemcpy(temp, temp_size_by_pre_id, sizeof(unsigned int)*num_parallel_blocks*target_N, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL({{pathobj}}_size_by_pre), &temp, sizeof(unsigned int*));
	int32_t* temp2;
	hipMalloc((void**)&temp2, sizeof(int32_t*)*num_parallel_blocks*target_N);
	hipMemcpy(temp2, temp_synapses_by_pre_id, sizeof(int32_t*)*num_parallel_blocks*target_N, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL({{pathobj}}_synapses_id_by_pre), &temp2, sizeof(int32_t**));
	{% endif %}

	unsigned int num_threads = max_delay;
	if(num_threads >= max_threads_per_block)
	{
		num_threads = max_threads_per_block;
	}
	_run_{{codeobj_name}}_kernel<<<1, num_threads>>>(
		source_N,
		num_parallel_blocks,
		max_threads_per_block,
		dt,
		syn_N,
		max_delay);

	//delete temp arrays
	delete [] h_synapses_synaptic_sources;
	delete [] h_synapses_synaptic_targets;
	delete [] h_synapses_delay;
	delete [] h_synapses_by_pre_id;
	delete [] h_delay_by_pre_id;
	delete [] temp_size_by_pre_id;
	delete [] temp_synapses_by_pre_id;
	delete [] temp_delay_by_pre_id;
}

{% endmacro %}

{% macro h_file() %}
#ifndef _INCLUDED_{{codeobj_name}}
#define _INCLUDED_{{codeobj_name}}

void _run_{{codeobj_name}}();

#endif
{% endmacro %}
