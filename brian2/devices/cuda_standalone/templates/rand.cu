{% macro cu_file() %}

#include "objects.h"
#include "synapses_classes.h"
#include "brianlib/clocks.h"
#include "network.h"
#include<iostream>
#include<fstream>
#include <hiprand.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

void _run_random_number_generation()
{
	using namespace brian;

	float mean = 0.0;
	float std_deviation = 1.0;

	{% for co in code_objects %}
	{% if co.rand_calls > 0 and co.runs_every_tick == True %}
	hiprandGenerateUniform(random_float_generator, dev_{{co.name}}_random_uniform_floats, {{co.owner._N}} * {{co.rand_calls}});
	{% endif %}
	{% if co.randn_calls > 0 and co.runs_every_tick == True %}
	hiprandGenerateNormal(random_float_generator, dev_{{co.name}}_random_normal_floats, {{co.owner._N}} * {{co.randn_calls}}, mean, std_deviation);
	{% endif %}
	{% endfor %}
}
{% endmacro %}

/////////////////////////////////////////////////////////////////////////////////////////////////////

{% macro h_file() %}

#ifndef _BRIAN_RAND_H
#define _BRIAN_RAND_H

#include <hiprand.h>

void _run_random_number_generation();

#endif


{% endmacro %}
