#include "hip/hip_runtime.h"
{% macro cu_file() %}

#include<stdint.h>
#include<vector>
#include "objects.h"
#include "synapses_classes.h"
#include "brianlib/clocks.h"
#include "brianlib/dynamic_array.h"
#include "network.h"
#include<iostream>
#include<fstream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand.h>

//////////////// clocks ///////////////////
{% for clock in clocks | sort(attribute='name') %}
Clock brian::{{clock.name}}({{clock.dt_}});
{% endfor %}

//////////////// networks /////////////////
{% for net in networks | sort(attribute='name') %}
Network brian::{{net.name}};
{% endfor %}

//////////////// arrays ///////////////////
{% for var, varname in array_specs | dictsort(by='value') %}
{% if not var in dynamic_array_specs %}
{{c_data_type(var.dtype)}} * brian::{{varname}};
{{c_data_type(var.dtype)}} * brian::dev{{varname}};
const int brian::_num_{{varname}} = {{var.size}};

{% endif %}
{% endfor %}

//////////////// dynamic arrays 1d /////////
{% for var, varname in dynamic_array_specs | dictsort(by='value') %}
std::vector<{{c_data_type(var.dtype)}}> brian::{{varname}};
thrust::device_vector<{{c_data_type(var.dtype)}}> brian::dev{{varname}};
{% endfor %}

//////////////// dynamic arrays 2d /////////
{% for var, varname in dynamic_array_2d_specs | dictsort(by='value') %}
thrust::device_vector<{{c_data_type(var.dtype)}}>* brian::{{varname}};
{% endfor %}

/////////////// static arrays /////////////
{% for (name, dtype_spec, N, filename) in static_array_specs | sort %}
{# arrays that are initialized from static data are already declared #}
{% if not name in array_specs.values() %}
{{dtype_spec}} * brian::{{name}};
{{dtype_spec}} * brian::dev{{name}};
const int brian::_num_{{name}} = {{N}};
{% endif %}
{% endfor %}

//////////////// synapses /////////////////
thrust::device_vector<int32_t> brian::synapses_by_pre_neuron;
{% for S in synapses | sort(attribute='name') %}
// {{S.name}}
Synapses<double> brian::{{S.name}}({{S.source|length}}, {{S.target|length}});
{% for path in S._pathways | sort(attribute='name') %}
// {{path.name}}
unsigned* brian::{{path.name}}_size_by_pre;
int32_t** brian::{{path.name}}_synapses_id_by_pre;
int32_t** brian::{{path.name}}_post_neuron_by_pre;
unsigned int** brian::{{path.name}}_delay_by_pre;
__device__ SynapticPathway<double> brian::{{path.name}};
{% endfor %}
{% endfor %}

unsigned int brian::num_cuda_processors;
unsigned int brian::max_threads_per_block;
unsigned int brian::max_shared_mem_size;

{% for S in synapses | sort(attribute='name') %}
{% for path in S._pathways | sort(attribute='name') %}
__global__ void {{path.name}}_init(
				unsigned int Nsource,
				unsigned int Ntarget,
				double* delays,
				int32_t* sources,
				int32_t* targets,
				double dt,
				int32_t start,
				int32_t stop
		)
{
	using namespace brian;

	{{path.name}}.init(Nsource, Ntarget, delays, sources, targets, dt, start, stop);
}
{% endfor %}
{% endfor %}

//////////////random numbers//////////////////
float* brian::dev_array_random_normal_floats;
float* brian::dev_array_random_uniform_floats;
__device__ float* brian::_array_randn;
__device__ float* brian::_array_rand;
unsigned int brian::num_random_normal_numbers;
unsigned int brian::num_random_uniform_numbers;
hiprandGenerator_t brian::random_float_generator;

void _init_arrays()
{
	using namespace brian;

	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, 0);

	num_cuda_processors = props.multiProcessorCount;
	max_threads_per_block = props.maxThreadsPerBlock;
	max_shared_mem_size = props.sharedMemPerBlock;

	num_random_normal_numbers = %RANDOM_NUMBER_NORMAL%;
	num_random_uniform_numbers = %RANDOM_NUMBER_UNIFORM%;
	hipMalloc((void**)&dev_array_random_normal_floats, sizeof(float)*num_random_normal_numbers);
	if(!dev_array_random_normal_floats)
	{
		printf("ERROR while allocating device memory with size %ld\n", sizeof(float)*num_random_normal_numbers);
	}
	hipMalloc((void**)&dev_array_random_uniform_floats, sizeof(float)*num_random_uniform_numbers);
	if(!dev_array_random_uniform_floats)
	{
		printf("ERROR while allocating device memory with size %ld\n", sizeof(float)*num_random_uniform_numbers);
	}
	hipMemcpyToSymbol(HIP_SYMBOL(_array_randn), dev_array_random_normal_floats, sizeof(float*));
	hipMemcpyToSymbol(HIP_SYMBOL(_array_rand), dev_array_random_uniform_floats, sizeof(float*));
	hiprandCreateGenerator(&random_float_generator, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(random_float_generator, time(0));
	hiprandGenerateNormal(random_float_generator, dev_array_random_normal_floats, 1, 0.0, 1.0);	//generating one number here fixes some hiprand bugs

	{% for S in synapses | sort(attribute='name') %}
	{% for path in S._pathways | sort(attribute='name') %}
	{{path.name}}_init<<<1,1>>>(
			{{path.source|length}},
			{{path.target|length}},
			thrust::raw_pointer_cast(&dev{{dynamic_array_specs[path.variables['delay']]}}[0]),
			thrust::raw_pointer_cast(&dev{{dynamic_array_specs[path.synapse_sources]}}[0]),
			thrust::raw_pointer_cast(&dev{{dynamic_array_specs[path.synapse_targets]}}[0]),
			{{path.source.dt_}},
			{{path.source.start}},
			{{path.source.stop}}
			);
	{% endfor %}
	{% endfor %}

    // Arrays initialized to 0
	{% for var in zero_arrays | sort(attribute='name') %}
	{% set varname = array_specs[var] %}
	{{varname}} = new {{c_data_type(var.dtype)}}[{{var.size}}];
	for(int i=0; i<{{var.size}}; i++) {{varname}}[i] = 0;
	hipMalloc((void**)&dev{{varname}}, sizeof({{c_data_type(var.dtype)}})*_num_{{varname}});
	if(!dev{{varname}})
	{
		printf("ERROR while allocating device memory with size %ld\n", sizeof({{c_data_type(var.dtype)}})*_num_{{varname}});
	}
	hipMemcpy(dev{{varname}}, {{varname}}, sizeof({{c_data_type(var.dtype)}})*_num_{{varname}}, hipMemcpyHostToDevice);

	{% endfor %}

	// Arrays initialized to an "arange"
	{% for var, start in arange_arrays %}
	{% set varname = array_specs[var] %}
	{{varname}} = new {{c_data_type(var.dtype)}}[{{var.size}}];
	for(int i=0; i<{{var.size}}; i++) {{varname}}[i] = {{start}} + i;
	hipMalloc((void**)&dev{{varname}}, sizeof({{c_data_type(var.dtype)}})*_num_{{varname}});
	if(!dev{{varname}})
	{
		printf("ERROR while allocating device memory with size %ld\n", sizeof({{c_data_type(var.dtype)}})*_num_{{varname}});
	}
	hipMemcpy(dev{{varname}}, {{varname}}, sizeof({{c_data_type(var.dtype)}})*_num_{{varname}}, hipMemcpyHostToDevice);

	{% endfor %}

	// static arrays
	{% for (name, dtype_spec, N, filename) in static_array_specs | sort %}
	{{name}} = new {{dtype_spec}}[{{N}}];
	hipMalloc((void**)&dev{{name}}, sizeof({{dtype_spec}})*{{N}});
	if(!dev{{name}})
	{
		printf("ERROR while allocating device memory with size %ld\n", sizeof({{dtype_spec}})*{{N}});
	}
	{% endfor %}

	{% for var, varname in dynamic_array_2d_specs | dictsort(by='value') %}
	{{varname}} = new thrust::device_vector<{{c_data_type(var.dtype)}}>[_num__array_statemonitor__indices];
	{% endfor %}
}

void _load_arrays()
{
	using namespace brian;

	{% for (name, dtype_spec, N, filename) in static_array_specs | sort %}
	ifstream f{{name}};
	f{{name}}.open("static_arrays/{{name}}", ios::in | ios::binary);
	if(f{{name}}.is_open())
	{
		f{{name}}.read(reinterpret_cast<char*>({{name}}), {{N}}*sizeof({{dtype_spec}}));
	} else
	{
		std::cout << "Error opening static array {{name}}." << endl;
	}
	{% endfor %}
}	

void _write_arrays()
{
	/*
	using namespace brian;

	{% for var, varname in array_specs | dictsort(by='value') %}
	{% if not (var in dynamic_array_specs or var in dynamic_array_2d_specs) %}
	ofstream outfile_{{varname}};
	outfile_{{varname}}.open("results/{{varname}}", ios::binary | ios::out);
	if(outfile_{{varname}}.is_open())
	{
		outfile_{{varname}}.write(reinterpret_cast<char*>({{varname}}), {{var.size}}*sizeof({{varname}}[0]));
		outfile_{{varname}}.close();
	} else
	{
		std::cout << "Error writing output file for {{varname}}." << endl;
	}
	{% endif %}
	{% endfor %}

	{% for var, varname in dynamic_array_specs | dictsort(by='value') %}
	ofstream outfile_{{varname}};
	outfile_{{varname}}.open("results/{{varname}}", ios::binary | ios::out);
	if(outfile_{{varname}}.is_open())
	{
		outfile_{{varname}}.write(reinterpret_cast<char*>(&{{varname}}[0]), {{varname}}.size()*sizeof({{varname}}[0]));
		outfile_{{varname}}.close();
	} else
	{
		std::cout << "Error writing output file for {{varname}}." << endl;
	}
	{% endfor %}

	{% for var, varname in dynamic_array_2d_specs | dictsort(by='value') %}
	ofstream outfile_{{varname}};
	outfile_{{varname}}.open("results/{{varname}}", ios::binary | ios::out);
	if(outfile_{{varname}}.is_open())
	{
        for (int n=0; n<{{varname}}.n; n++)
        {
            outfile_{{varname}}.write(reinterpret_cast<char*>(&{{varname}}(n, 0)), {{varname}}.m*sizeof({{varname}}(0, 0)));
        }
        outfile_{{varname}}.close();
	} else
	{
		std::cout << "Error writing output file for {{varname}}." << endl;
	}
	{% endfor %}
	*/
}

{% for S in synapses | sort(attribute='name') %}
{% for path in S._pathways | sort(attribute='name') %}
__global__ void {{path.name}}_destroy()
{
	using namespace brian;

	{{path.name}}.destroy();
}
{% endfor %}
{% endfor %}

void _dealloc_arrays()
{
	using namespace brian;

	{% for S in synapses | sort(attribute='name') %}
	{% for path in S._pathways | sort(attribute='name') %}
	{{path.name}}_destroy<<<1,1>>>();
	{% endfor %}
	{% endfor %}

	{% for var, varname in dynamic_array_specs | dictsort(by='value') %}
	dev{{varname}}.clear();
	thrust::device_vector<{{c_data_type(var.dtype)}}>().swap(dev{{varname}});
	{% endfor %}

	{% for var, varname in array_specs | dictsort(by='value') %}
	{% if not var in dynamic_array_specs %}
	if({{varname}}!=0)
	{
		delete [] {{varname}};
		{{varname}} = 0;
	}
	if(dev{{varname}}!=0)
	{
		hipFree(dev{{varname}});
		dev{{varname}} = 0;
	}
	{% endif %}
	{% endfor %}

	// static arrays
	{% for (name, dtype_spec, N, filename) in static_array_specs | sort %}
	if({{name}}!=0)
	{
		delete [] {{name}};
		{{name}} = 0;
	}
	{% endfor %}
}

{% endmacro %}

/////////////////////////////////////////////////////////////////////////////////////////////////////

{% macro h_file() %}

#ifndef _BRIAN_OBJECTS_H
#define _BRIAN_OBJECTS_H

#include<vector>
#include<stdint.h>
#include "synapses_classes.h"
#include "brianlib/clocks.h"
#include "brianlib/dynamic_array.h"
#include "network.h"

#include <thrust/device_vector.h>
#include <hiprand.h>

namespace brian {

//////////////// clocks ///////////////////
{% for clock in clocks %}
extern Clock {{clock.name}};
{% endfor %}

//////////////// networks /////////////////
extern Network magicnetwork;
{% for net in networks %}
extern Network {{net.name}};
{% endfor %}

//////////////// dynamic arrays ///////////
{% for var, varname in dynamic_array_specs | dictsort(by='value') %}
extern std::vector<{{c_data_type(var.dtype)}}> {{varname}};
extern thrust::device_vector<{{c_data_type(var.dtype)}}> dev{{varname}};
{% endfor %}

//////////////// arrays ///////////////////
{% for var, varname in array_specs | dictsort(by='value') %}
{% if not var in dynamic_array_specs %}
extern {{c_data_type(var.dtype)}} * {{varname}};
extern {{c_data_type(var.dtype)}} * dev{{varname}};
extern const int _num_{{varname}};
{% endif %}
{% endfor %}

//////////////// dynamic arrays 2d /////////
{% for var, varname in dynamic_array_2d_specs | dictsort(by='value') %}
extern thrust::device_vector<{{c_data_type(var.dtype)}}>* {{varname}};
{% endfor %}

/////////////// static arrays /////////////
{% for (name, dtype_spec, N, filename) in static_array_specs | sort %}
{# arrays that are initialized from static data are already declared #}
{% if not name in array_specs.values() %}
extern {{dtype_spec}} *{{name}};
extern {{dtype_spec}} *dev{{name}};
extern const int _num_{{name}};
{% endif %}
{% endfor %}

//////////////// synapses /////////////////
extern thrust::device_vector<int32_t> synapses_by_pre_neuron;	//in CSR format: neuron 0 has syns from arr[0] to (arr[1] - 1), neuron 1 from arr[1] to (arr[2] - 1), etc...
{% for S in synapses | sort(attribute='name') %}
// {{S.name}}
extern Synapses<double> {{S.name}};
{% for path in S._pathways | sort(attribute='name') %}
extern unsigned* {{path.name}}_size_by_pre;
extern int32_t** {{path.name}}_synapses_id_by_pre;
extern int32_t** {{path.name}}_post_neuron_by_pre;
extern unsigned int** {{path.name}}_delay_by_pre;
extern __device__ SynapticPathway<double> {{path.name}};
{% endfor %}
{% endfor %}

//////////////// random numbers /////////////////
extern float* dev_array_random_normal_floats;
extern float* dev_array_random_uniform_floats;
extern __device__ float* _array_randn;
extern __device__ float* _array_rand;
extern unsigned int num_random_normal_numbers;
extern unsigned int num_random_uniform_numbers;
extern hiprandGenerator_t random_float_generator;

//CUDA
extern unsigned int num_cuda_processors;
extern unsigned int max_threads_per_block;
extern unsigned int max_shared_mem_size;

}

void _init_arrays();
void _load_arrays();
void _write_arrays();
void _dealloc_arrays();

#endif


{% endmacro %}
