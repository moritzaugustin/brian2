#include "hip/hip_runtime.h"
{% macro cu_file() %}

#include<stdint.h>
#include<vector>
#include "objects.h"
#include "synapses_classes.h"
#include "brianlib/clocks.h"
#include "brianlib/dynamic_array.h"
#include "network.h"
#include<iostream>
#include<fstream>
#include <thrust/device_vector.h>

//////////////// clocks ///////////////////
{% for clock in clocks | sort(attribute='name') %}
Clock brian::{{clock.name}}({{clock.dt_}});
{% endfor %}

//////////////// networks /////////////////
{% for net in networks | sort(attribute='name') %}
Network brian::{{net.name}};
{% endfor %}

//////////////// arrays ///////////////////
{% for var, varname in array_specs | dictsort(by='value') %}
{% if not var in dynamic_array_specs %}
{{c_data_type(var.dtype)}} * brian::{{varname}};
{{c_data_type(var.dtype)}} * brian::dev{{varname}};
const int brian::_num_{{varname}} = {{var.size}};

{% endif %}
{% endfor %}

//////////////// dynamic arrays 1d /////////
{% for var, varname in dynamic_array_specs | dictsort(by='value') %}
std::vector<{{c_data_type(var.dtype)}}> brian::{{varname}};
thrust::device_vector<{{c_data_type(var.dtype)}}> brian::dev{{varname}};
{% endfor %}

//////////////// dynamic arrays 2d /////////
{% for var, varname in dynamic_array_2d_specs | dictsort(by='value') %}
DynamicArray2D<{{c_data_type(var.dtype)}}> brian::{{varname}};
{% endfor %}

/////////////// static arrays /////////////
{% for (name, dtype_spec, N, filename) in static_array_specs | sort %}
{# arrays that are initialized from static data are already declared #}
{% if not name in array_specs.values() %}
{{dtype_spec}} * brian::{{name}};
{{dtype_spec}} * brian::dev{{name}};
const int brian::_num_{{name}} = {{N}};
{% endif %}
{% endfor %}

//////////////// synapses /////////////////
{% for S in synapses | sort(attribute='name') %}
// {{S.name}}
Synapses<double> brian::{{S.name}}({{S.source|length}}, {{S.target|length}});
{% for path in S._pathways | sort(attribute='name') %}
SynapticPathway<double> brian::{{path.name}}(
		{{path.source|length}}, {{path.target|length}},
		{{dynamic_array_specs[path.variables['delay']]}},
		{{dynamic_array_specs[path.synapse_sources]}},
		{{path.source.dt_}},
		{{path.source.start}}, {{path.source.stop}});
{% endfor %}
{% endfor %}

unsigned int brian::num_cuda_processors;
unsigned int brian::max_threads_per_block;
unsigned int brian::max_shared_mem_size;


void _init_arrays()
{
	using namespace brian;

	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, 0);

	num_cuda_processors = props.multiProcessorCount;
	max_threads_per_block = props.maxThreadsPerBlock;
	max_shared_mem_size = props.sharedMemPerBlock;

    // Arrays initialized to 0
	{% for var in zero_arrays | sort(attribute='name') %}
	{% set varname = array_specs[var] %}
	{{varname}} = new {{c_data_type(var.dtype)}}[{{var.size}}];
	for(int i=0; i<{{var.size}}; i++) {{varname}}[i] = 0;
	hipMalloc((void**)&dev{{varname}}, sizeof({{c_data_type(var.dtype)}})*_num_{{varname}});
	if(!dev{{varname}})
	{
		printf("ERROR while allocating device memory with size %ld\n", sizeof({{c_data_type(var.dtype)}})*_num_{{varname}});
	}
	hipMemcpy(dev{{varname}}, {{varname}}, sizeof({{c_data_type(var.dtype)}})*_num_{{varname}}, hipMemcpyHostToDevice);

	{% endfor %}

	// Arrays initialized to an "arange"
	{% for var, start in arange_arrays %}
	{% set varname = array_specs[var] %}
	{{varname}} = new {{c_data_type(var.dtype)}}[{{var.size}}];
	for(int i=0; i<{{var.size}}; i++) {{varname}}[i] = {{start}} + i;
	hipMalloc((void**)&dev{{varname}}, sizeof({{c_data_type(var.dtype)}})*_num_{{varname}});
	if(!dev{{varname}})
	{
		printf("ERROR while allocating device memory with size %ld\n", sizeof({{c_data_type(var.dtype)}})*_num_{{varname}});
	}
	hipMemcpy(dev{{varname}}, {{varname}}, sizeof({{c_data_type(var.dtype)}})*_num_{{varname}}, hipMemcpyHostToDevice);

	{% endfor %}

	// static arrays
	{% for (name, dtype_spec, N, filename) in static_array_specs | sort %}
	{{name}} = new {{dtype_spec}}[{{N}}];
	hipMalloc((void**)&dev{{name}}, sizeof({{dtype_spec}})*{{N}});
	if(!dev{{name}})
	{
		printf("ERROR while allocating device memory with size %ld\n", sizeof({{dtype_spec}})*{{N}});
	}

	{% endfor %}
}

void _load_arrays()
{
	using namespace brian;

	{% for (name, dtype_spec, N, filename) in static_array_specs | sort %}
	ifstream f{{name}};
	f{{name}}.open("static_arrays/{{name}}", ios::in | ios::binary);
	if(f{{name}}.is_open())
	{
		f{{name}}.read(reinterpret_cast<char*>({{name}}), {{N}}*sizeof({{dtype_spec}}));
	} else
	{
		std::cout << "Error opening static array {{name}}." << endl;
	}
	{% endfor %}
}	

void _write_arrays()
{
	using namespace brian;

	{% for var, varname in array_specs | dictsort(by='value') %}
	{% if not (var in dynamic_array_specs or var in dynamic_array_2d_specs) %}
	ofstream outfile_{{varname}};
	outfile_{{varname}}.open("results/{{varname}}", ios::binary | ios::out);
	if(outfile_{{varname}}.is_open())
	{
		outfile_{{varname}}.write(reinterpret_cast<char*>({{varname}}), {{var.size}}*sizeof({{varname}}[0]));
		outfile_{{varname}}.close();
	} else
	{
		std::cout << "Error writing output file for {{varname}}." << endl;
	}
	{% endif %}
	{% endfor %}

	{% for var, varname in dynamic_array_specs | dictsort(by='value') %}
	ofstream outfile_{{varname}};
	outfile_{{varname}}.open("results/{{varname}}", ios::binary | ios::out);
	if(outfile_{{varname}}.is_open())
	{
		outfile_{{varname}}.write(reinterpret_cast<char*>(&{{varname}}[0]), {{varname}}.size()*sizeof({{varname}}[0]));
		outfile_{{varname}}.close();
	} else
	{
		std::cout << "Error writing output file for {{varname}}." << endl;
	}
	{% endfor %}

	{% for var, varname in dynamic_array_2d_specs | dictsort(by='value') %}
	ofstream outfile_{{varname}};
	outfile_{{varname}}.open("results/{{varname}}", ios::binary | ios::out);
	if(outfile_{{varname}}.is_open())
	{
        for (int n=0; n<{{varname}}.n; n++)
        {
            outfile_{{varname}}.write(reinterpret_cast<char*>(&{{varname}}(n, 0)), {{varname}}.m*sizeof({{varname}}(0, 0)));
        }
        outfile_{{varname}}.close();
	} else
	{
		std::cout << "Error writing output file for {{varname}}." << endl;
	}
	{% endfor %}
}

void _dealloc_arrays()
{
	using namespace brian;


	{% for var, varname in dynamic_array_specs | dictsort(by='value') %}
	dev{{varname}}.clear();
	thrust::device_vector<{{c_data_type(var.dtype)}}>().swap(dev{{varname}});
	{% endfor %}

	{% for var, varname in array_specs | dictsort(by='value') %}
	{% if not var in dynamic_array_specs %}
	if({{varname}}!=0)
	{
		delete [] {{varname}};
		{{varname}} = 0;
	}
	if(dev{{varname}}!=0)
	{
		hipFree(dev{{varname}});
		dev{{varname}} = 0;
	}
	{% endif %}
	{% endfor %}

	// static arrays
	{% for (name, dtype_spec, N, filename) in static_array_specs | sort %}
	if({{name}}!=0)
	{
		delete [] {{name}};
		{{name}} = 0;
	}
	{% endfor %}
}

{% endmacro %}

/////////////////////////////////////////////////////////////////////////////////////////////////////

{% macro h_file() %}

#ifndef _BRIAN_OBJECTS_H
#define _BRIAN_OBJECTS_H

#include<vector>
#include<stdint.h>
#include "synapses_classes.h"
#include "brianlib/clocks.h"
#include "brianlib/dynamic_array.h"
#include "network.h"

#include <thrust/device_vector.h>

namespace brian {

//////////////// clocks ///////////////////
{% for clock in clocks %}
extern Clock {{clock.name}};
{% endfor %}

//////////////// networks /////////////////
extern Network magicnetwork;
{% for net in networks %}
extern Network {{net.name}};
{% endfor %}

//////////////// dynamic arrays ///////////
{% for var, varname in dynamic_array_specs | dictsort(by='value') %}
extern std::vector<{{c_data_type(var.dtype)}}> {{varname}};
extern thrust::device_vector<{{c_data_type(var.dtype)}}> dev{{varname}};
{% endfor %}

//////////////// arrays ///////////////////
{% for var, varname in array_specs | dictsort(by='value') %}
{% if not var in dynamic_array_specs %}
extern {{c_data_type(var.dtype)}} * {{varname}};
extern {{c_data_type(var.dtype)}} * dev{{varname}};
extern const int _num_{{varname}};
{% endif %}
{% endfor %}

//////////////// dynamic arrays 2d /////////
{% for var, varname in dynamic_array_2d_specs | dictsort(by='value') %}
extern DynamicArray2D<{{c_data_type(var.dtype)}}> {{varname}};
{% endfor %}

/////////////// static arrays /////////////
{% for (name, dtype_spec, N, filename) in static_array_specs | sort %}
{# arrays that are initialized from static data are already declared #}
{% if not name in array_specs.values() %}
extern {{dtype_spec}} *{{name}};
extern {{dtype_spec}} *dev{{name}};
extern const int _num_{{name}};
{% endif %}
{% endfor %}

//////////////// synapses /////////////////
{% for S in synapses | sort(attribute='name') %}
// {{S.name}}
extern Synapses<double> {{S.name}};
{% for path in S._pathways | sort(attribute='name') %}
extern SynapticPathway<double> {{path.name}};
{% endfor %}
{% endfor %}

extern unsigned int num_cuda_processors;
extern unsigned int max_threads_per_block;
extern unsigned int max_shared_mem_size;

}

void _init_arrays();
void _load_arrays();
void _write_arrays();
void _dealloc_arrays();

#endif


{% endmacro %}
