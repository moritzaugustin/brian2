#include "hip/hip_runtime.h"
{% extends 'common_synapses.cu' %}

{% set _non_synaptic = [] %}
{% for var in variables %}
    {% if variable_indices[var] != '_idx' %}
        {# This is a trick to get around the scoping problem #}
        {% if _non_synaptic.append(1) %}{% endif %}
    {% endif %}
{% endfor %}

{% block kernel %}

__global__ void kernel_{{codeobj_name}}(
	unsigned int bid_offset,
	unsigned int THREADS_PER_BLOCK,
	%DEVICE_PARAMETERS%
	)
{
	{# USES_VARIABLES { N, _synaptic_pre, _spikespace} #}
	using namespace brian;

	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x + bid_offset;
	unsigned int _idx = bid * THREADS_PER_BLOCK + tid;
	unsigned int _vectorisation_idx = _idx;
	%KERNEL_VARIABLES%
	{% block additional_variables %}
	{% endblock %}

	cudaVector<int32_t>* synapses_queue;
	
	{{pathway.name}}.queue->peek(
		&synapses_queue);

	if(tid == 0 && bid == 0) printf("START SYN EFFECTS\n");

			
	{{scalar_code|autoindent}}

	if (!({{pathway.name}}.no_or_const_delay_mode))
	{
		int size = synapses_queue[bid].size();
		for(int j = tid; j < size; j+=THREADS_PER_BLOCK)
		{
			int32_t _idx = synapses_queue[bid].at(j);
	
			{{vector_code|autoindent}}
		}
	}
	else
	{
		if(bid != 0)
			return;
		//no or const delay mode
		for(int j = 0; j < _num_spikespace; j++)
		{
			int32_t spiking_neuron = {{_spikespace}}[j];
			if(spiking_neuron == -1)
			{
				break;
			}
			for(int i = tid; i < {{pathway.name}}_size_by_pre[spiking_neuron]; i+= THREADS_PER_BLOCK)
			{
				int32_t _idx = {{pathway.name}}_synapses_id_by_pre[spiking_neuron][i];
			
				{{vector_code|autoindent}}
			}
			__syncthreads();
		}
	}
}

{% endblock %}

{% block kernel_call %}
	{% if serializing_mode == "syn" %}
	kernel_{{codeobj_name}}<<<num_parallel_blocks,max_threads_per_block>>>(
		0,
		max_threads_per_block,
		%HOST_PARAMETERS%
	);
	{% endif %}
	{% if serializing_mode == "post" %}
	kernel_{{codeobj_name}}<<<num_parallel_blocks,1>>>(
		0,
		1,
		%HOST_PARAMETERS%
	);
	{% endif %}
	{% if serializing_mode == "pre" %}
	for(int i = 0; i < num_parallel_blocks; i++)
	{
		kernel_{{codeobj_name}}<<<1,1>>>(
			i,
			1,
			%HOST_PARAMETERS%
		);
	}
	{% endif %}
{% endblock %}

{% block extra_maincode %}
{% endblock %}

{% block extra_functions_cu %}
void _debugmsg_{{codeobj_name}}()
{
	using namespace brian;
	std::cout << "Number of synapses: " << dev{{_dynamic__synaptic_pre}}.size() << endl;
}
{% endblock %}

{% block extra_functions_h %}
void _debugmsg_{{codeobj_name}}();
{% endblock %}

{% macro main_finalise() %}
_debugmsg_{{codeobj_name}}();
{% endmacro %}
