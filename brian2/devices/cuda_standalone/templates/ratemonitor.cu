#include "hip/hip_runtime.h"
{% extends 'common_group.cu' %}
{# USES_VARIABLES { rate, t, _spikespace, _clock_t, _clock_dt,
                    _num_source_neurons, _source_start, _source_stop } #}

{% block extra_maincode %}
{{_dynamic_t}}.push_back(_clock_t);
{{_dynamic_rate}}.push_back(0.0);	//push dummy value

double* dev{{_dynamic_rate}} = thrust::raw_pointer_cast(&({{_dynamic_rate}}[0]));
int index_last_element = {{_dynamic_rate}}.size() - 1;
{% endblock %}

{% block kernel_call %}
_run_{{codeobj_name}}_kernel<<<1,1>>>(
	{{owner.source.N}},
	_clock_dt,
	index_last_element,
	dev_array_{{owner.source.name}}__spikespace,
	dev{{_dynamic_rate}});
{% endblock %}

{% block kernel %}
__global__ void _run_{{codeobj_name}}_kernel(
	unsigned int N,
	double _clock_dt,
	int32_t index_last_element,
	int32_t* spikespace,
	double* ratemonitor_rate
	)
{
	using namespace brian;

	unsigned int num_spikes = spikespace[N];
	ratemonitor_rate[index_last_element] = 1.0*num_spikes/_clock_dt/N;
}
{% endblock %}
