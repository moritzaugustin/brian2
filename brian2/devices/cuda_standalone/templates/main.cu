#include <stdlib.h>
#include "objects.h"
#include <ctime>
#include <time.h>
#include "run.h"
#include "brianlib/common_math.h"
#include "rand.h"

{% for codeobj in code_objects %}
#include "code_objects/{{codeobj.name}}.h"
{% endfor %}

{% for name in additional_headers %}
#include "{{name}}"
{% endfor %}

#include <iostream>
#include <fstream>

{{report_func|autoindent}}

int main(int argc, char **argv)
{	
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, 0);
	size_t limit = 32 * 1024 * 1024;
	hipDeviceSetLimit(hipLimitMallocHeapSize, limit);
	hipDeviceSynchronize();
	
	brian_start();

	{
		using namespace brian;

        {{main_lines|autoindent}}
	}

	brian_end();

	return 0;
}
