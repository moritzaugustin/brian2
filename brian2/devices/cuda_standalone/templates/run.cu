#include "hip/hip_runtime.h"
{% macro cu_file() %}
#include<stdlib.h>
#include "objects.h"
#include<ctime>

{% for codeobj in code_objects | sort(attribute='name') %}
#include "code_objects/{{codeobj.name}}.h"
{% endfor %}

{% for name in additional_headers %}
#include "{{name}}"
{% endfor %}

void _sync_clocks()
{
	using namespace brian;
	
    {% for clock in clocks | sort(attribute='name') %}
    hipMemcpy(dev{{array_specs[clock.variables['timestep']]}}, {{array_specs[clock.variables['timestep']]}}, sizeof(uint64_t)*_num_{{array_specs[clock.variables['timestep']]}}, hipMemcpyHostToDevice);
    hipMemcpy(dev{{array_specs[clock.variables['dt']]}}, {{array_specs[clock.variables['dt']]}}, sizeof(double)*_num_{{array_specs[clock.variables['dt']]}}, hipMemcpyHostToDevice);
    hipMemcpy(dev{{array_specs[clock.variables['t']]}}, {{array_specs[clock.variables['t']]}}, sizeof(double)*_num_{{array_specs[clock.variables['t']]}}, hipMemcpyHostToDevice);
    {% endfor %}
}

void brian_start()
{
	_init_arrays();
	_load_arrays();
	// Initialize clocks (link timestep and dt to the respective arrays)
    {% for clock in clocks | sort(attribute='name') %}
    brian::{{clock.name}}.timestep = brian::{{array_specs[clock.variables['timestep']]}};
    brian::{{clock.name}}.dt = brian::{{array_specs[clock.variables['dt']]}};
    brian::{{clock.name}}.t = brian::{{array_specs[clock.variables['t']]}};
    {% endfor %}
	srand((unsigned int)time(NULL));
}

void brian_end()
{
	_write_arrays();
	_dealloc_arrays();
}

{% for name, lines in run_funcs.items() | sort(attribute='name') %}
void {{name}}()
{
	using namespace brian;

    {{lines|autoindent}}
}

{% endfor %}

{% endmacro %}

/////////////////////////////////////////////////////////////////////////////////////////////////////

{% macro h_file() %}

void brian_start();
void brian_end();
void _sync_clocks();

{% for name, lines in run_funcs.items() | sort(attribute='name') %}
void {{name}}();
{% endfor %}

{% endmacro %}
