#include "hip/hip_runtime.h"
{% macro cu_file() %}
#include "code_objects/{{codeobj_name}}.h"
#include<math.h>
#include "brianlib/common_math.h"
#include<stdint.h>
#include<iostream>
#include<fstream>
{% block extra_headers %}
{% endblock %}

////// SUPPORT CODE ///////
namespace {
	int num_blocks(int objects){
		return ceil(objects / (double)brian::max_threads_per_block);
	}
	int num_threads(int objects){
		return brian::max_threads_per_block;
	}
	{% block extra_device_helper %}
	{% endblock %}
	{{support_code_lines|autoindent}}
}

{% block kernel %}}
__global__ void _kernel_{{codeobj_name}}(
	unsigned int THREADS_PER_BLOCK,
	%DEVICE_PARAMETERS%
	)
{
	{# USES_VARIABLES { N } #}
	using namespace brian;

	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	unsigned int _idx = bid * THREADS_PER_BLOCK + tid;
	unsigned int _vectorisation_idx = _idx;
	%KERNEL_VARIABLES%

	if(_idx < 0 || _idx >= N)
	{
		return;
	}

	{% block maincode %}
	{% block maincode_inner %}
	{{scalar_code|autoindent}}
	{{vector_code|autoindent}}
	{% endblock %}
	{% endblock %}
}
{% endblock %}

////// HASH DEFINES ///////
{{hashdefine_lines|autoindent}}

void _run_{{codeobj_name}}()
{	
	using namespace brian;
	///// CONSTANTS ///////////
	%CONSTANTS%

	{% block kernel_call %}
	kernel_{{codeobj_name}}<<<num_blocks(N),num_threads(N)>>>(
			num_threads(N),
			%HOST_PARAMETERS%
		);
	{% endblock %}
}

{% block extra_functions_cu %}
{% endblock %}

{% endmacro %}


{% macro h_file() %}
#ifndef _INCLUDED_{{codeobj_name}}
#define _INCLUDED_{{codeobj_name}}

#include "objects.h"

void _run_{{codeobj_name}}();

{% block extra_functions_h %}
{% endblock %}

#endif
{% endmacro %}
