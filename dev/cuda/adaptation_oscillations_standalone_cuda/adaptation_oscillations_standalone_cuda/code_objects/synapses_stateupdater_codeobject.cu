#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/synapses_stateupdater_codeobject.h"
#include<math.h>
#include "brianlib/common_math.h"
#include<stdint.h>
#include<iostream>
#include<fstream>

#define THREADS 1024
#define BLOCKS(N) (N + THREADS -1)/THREADS

/*
__global__ void _run_synapses_stateupdater_codeobject_kernel()
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;

	//DO NOTHING IN THIS KERNEL
}
*/

void _run_synapses_stateupdater_codeobject()
{
	using namespace brian;

	const int64_t N = synapses._N();

	//_run_synapses_group_variable_set_conditional_codeobject_kernel<<<BLOCKS(N), THREADS>>>();
}

