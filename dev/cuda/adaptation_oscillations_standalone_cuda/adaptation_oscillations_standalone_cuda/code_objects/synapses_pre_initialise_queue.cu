#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/synapses_pre_initialise_queue.h"

#define N 4000
#define ceil(M, num) ((M + num-1)/num)

__global__ void _run_synapses_pre_initialise_queue_kernel(double* par_real_delays, int32_t* par_sources, int32_t* par_targets, int32_t* par_pos, int n_synapses)
{
	using namespace brian;

	synapses_pre.queue->prepare(par_real_delays, par_sources, par_targets, par_pos, n_synapses, synapses_pre.dt);
}

void _run_synapses_pre_initialise_queue()
{
	using namespace brian;

	int syn_N = _dynamic_array_synapses__synaptic_pre.size();

	_run_synapses_pre_initialise_queue_kernel<<<1, 1>>>(
		thrust::raw_pointer_cast(&_dynamic_array_synapses_pre_delay[0]),
		thrust::raw_pointer_cast(&_dynamic_array_synapses__synaptic_pre[0]),
		thrust::raw_pointer_cast(&_dynamic_array_synapses__synaptic_post[0]),
		thrust::raw_pointer_cast(&_dynamic_array_synapses__pos[0]),
		syn_N);
}
