#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/neurongroup_group_variable_set_conditional_codeobject.h"
#include<math.h>
#include "brianlib/common_math.h"
#include<stdint.h>
#include<iostream>
#include<fstream>

__global__ void _run_neurongroup_group_variable_set_conditional_codeobject_kernel(
	unsigned int _neurongroup_N,
	unsigned int max_threads_per_block,
	float* _array_rands,
	double* _array_neurongroup_v,
	bool* _array_neurongroup_not_refractory)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;

	float* _ptr_array_rands = _array_rands;
	double* _ptr_array_neurongroup_v = _array_neurongroup_v;
	bool* _ptr_array_neurongroup_not_refractory = _array_neurongroup_not_refractory;
	
	int neuron_id = bid*max_threads_per_block + tid;
	if(neuron_id < 0 || neuron_id >= _neurongroup_N)
	{
		return;
	}

	bool _cond = true;
	if(_cond)
	{
		const bool not_refractory = _ptr_array_neurongroup_not_refractory[neuron_id];
		double v = _ptr_array_neurongroup_v[neuron_id];
		if(not_refractory)
		{
			float r = _ptr_array_rands[neuron_id];	//get random pregenerated number
			v = r * 0.001;
			_ptr_array_neurongroup_v[neuron_id] = v;
		}
	}
}

void _run_neurongroup_group_variable_set_conditional_codeobject()
{
	using namespace brian;

	//genenerate an arry of random numbers on the device
	float* dev_array_rands;
	hipMalloc((void**)&dev_array_rands, sizeof(float)*neurongroup_N);
	if(!dev_array_rands)
	{
		printf("ERROR while allocating device memory with size %ld in _run_neurongroup_group_variable_set_conditional_codeobject()\n", sizeof(float)*neurongroup_N);
	}
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, time(0));
	hiprandGenerateUniform(gen, dev_array_rands, neurongroup_N);

	unsigned int blocks = (neurongroup_N + max_threads_per_block - 1)/max_threads_per_block;	// = ceil(N/num_threads)

	_run_neurongroup_group_variable_set_conditional_codeobject_kernel<<<blocks, max_threads_per_block>>>(
		neurongroup_N,
		max_threads_per_block,
		dev_array_rands,
		dev_array_neurongroup_v,
		dev_array_neurongroup_not_refractory);

	hiprandDestroyGenerator(gen);
	hipFree(dev_array_rands);
}


