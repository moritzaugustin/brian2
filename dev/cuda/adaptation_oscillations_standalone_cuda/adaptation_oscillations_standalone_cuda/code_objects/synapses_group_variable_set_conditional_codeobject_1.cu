#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/synapses_group_variable_set_conditional_codeobject_1.h"
#include<math.h>
#include "brianlib/common_math.h"
#include<stdint.h>
#include<iostream>
#include<fstream>

__global__ void _run_synapses_group_variable_set_conditional_codeobject_1_kernel(
	int _syn_N,
	unsigned int max_threads_per_block,
	double* _array_synapses_pre_delay,
	float* _array_rands)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;

	int syn_N = _syn_N;
	double* _ptr_array_synapses_pre_delay = _array_synapses_pre_delay;
	float* _ptr_array_rands = _array_rands;

	int syn_id = bid*max_threads_per_block + tid;
	if(syn_id < 0 || syn_id >= syn_N)
	{
		return;
	}

	const bool _cond = true;
	if(_cond)
	{
		double delay;
		float r = _ptr_array_rands[syn_id];
		delay = 0.002 + r * 0.001 - 0.001 / 2;
		_ptr_array_synapses_pre_delay[syn_id] = delay;
	}
}

void _run_synapses_group_variable_set_conditional_codeobject_1()
{
	using namespace brian;

	const int syn_N = synapses._N();
	double* const dev_array_synapses_pre_delay = thrust::raw_pointer_cast(&_dynamic_array_synapses_pre_delay[0]);

	//genenerate an arry of random numbers on the device
	float* dev_array_rands;
	hipMalloc((void**)&dev_array_rands, sizeof(float)*syn_N);
	if(!dev_array_rands)
	{
		printf("ERROR while allocating device memory with size %ld in _run_synapses_group_variable_set_conditional_codeobject_1()\n", sizeof(float)*syn_N);
	}
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, time(0));
	hiprandGenerateUniform(gen, dev_array_rands, syn_N);

	unsigned int blocks = (syn_N + max_threads_per_block - 1)/max_threads_per_block;	// = ceil(N/num_threads)

	_run_synapses_group_variable_set_conditional_codeobject_1_kernel<<<blocks, max_threads_per_block>>>(
		syn_N,
		max_threads_per_block,
		dev_array_synapses_pre_delay,
		dev_array_rands);

	hiprandDestroyGenerator(gen);
	hipFree(dev_array_rands);
}

