#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/neurongroup_group_variable_set_conditional_codeobject_1.h"
#include<math.h>
#include "brianlib/common_math.h"
#include<stdint.h>
#include<iostream>
#include<fstream>

#define neuron_N 4000
#define THREADS 1024
#define BLOCKS (neuron_N + THREADS -1)/THREADS

__global__ void _run_neurongroup_group_variable_set_conditional_codeobject_1_kernel(
	float* par_array_rands,
	double* par_array_neurongroup_w,
	bool* par_array_neurongroup_not_refractory)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;

	float* _ptr_array_rands = par_array_rands;
	double* _ptr_array_neurongroup_w = par_array_neurongroup_w;
	bool* _ptr_array_neurongroup_not_refractory = par_array_neurongroup_not_refractory;
	
	int neuron_id = bid*THREADS + tid;
	if(neuron_id < 0 || neuron_id >= neuron_N)
	{
		return;
	}

	bool _cond = true;
	if(_cond)
	{
		const bool not_refractory = _ptr_array_neurongroup_not_refractory[neuron_id];
		double w = _ptr_array_neurongroup_w[neuron_id];
		if(not_refractory)
		{
			float r = _ptr_array_rands[neuron_id];	//get random pregenerated number
			w = r * 10 * 0.0001;
			_ptr_array_neurongroup_w[neuron_id] = w;
		}
	}
}

void _run_neurongroup_group_variable_set_conditional_codeobject_1()
{
	using namespace brian;

	float* dev_array_rands;
	hipMalloc((void**)&dev_array_rands, sizeof(float)*neuron_N);
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, time(0));
	hiprandGenerateUniform(gen, dev_array_rands, neuron_N);

	_run_neurongroup_group_variable_set_conditional_codeobject_1_kernel<<<BLOCKS, THREADS>>>(
		dev_array_rands,
		dev_array_neurongroup_w,
		dev_array_neurongroup_not_refractory);

	hiprandDestroyGenerator(gen);
	hipFree(dev_array_rands);
}

