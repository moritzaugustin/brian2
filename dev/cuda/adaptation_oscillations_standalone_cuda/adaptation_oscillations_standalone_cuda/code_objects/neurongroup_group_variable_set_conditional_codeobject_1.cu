#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/neurongroup_group_variable_set_conditional_codeobject_1.h"
#include<math.h>
#include "brianlib/common_math.h"
#include<stdint.h>
#include<iostream>
#include<fstream>

#define N 4000
#define ceil(N, num) ((N + num-1)/num)

__global__ void _run_neurongroup_group_variable_set_conditional_codeobject_kernel_1(int par_N, double* par_array_neurongroup_w, bool* par_array_neurongroup_not_refractory, float* par_rands)
{
	using namespace brian;

	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int idx = bid * 1024 + tid;

	if(idx >= par_N)
	{
		return;
	}

	double * _ptr_array_neurongroup_w= par_array_neurongroup_w;
	bool * _ptr_array_neurongroup_not_refractory = par_array_neurongroup_not_refractory;
	float r = par_rands[idx];

	double w = _ptr_array_neurongroup_w[idx];
	bool not_refractory = _ptr_array_neurongroup_not_refractory[idx];
	bool _cond = true;
	if(_cond)
	{
		if(not_refractory)
		{
			w = r * 10 * 0.0001;
		}
		_ptr_array_neurongroup_w[idx] = w;
	}
}


void _run_neurongroup_group_variable_set_conditional_codeobject_1()
{
	using namespace brian;

	float* dev_array_rands;
	hipMalloc((void**)&dev_array_rands, sizeof(float)*N);
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, time(0));
	hiprandGenerateUniform(gen, dev_array_rands, N);

	_run_neurongroup_group_variable_set_conditional_codeobject_kernel_1<<<ceil(N, 1024),1024>>>(N, dev_array_neurongroup_w, dev_array_neurongroup_not_refractory, dev_array_rands);

	hiprandDestroyGenerator(gen);
	hipFree(dev_array_rands);
}

