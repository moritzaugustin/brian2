#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/neurongroup_group_variable_set_conditional_codeobject_1.h"
#include<math.h>
#include "brianlib/common_math.h"
#include<stdint.h>
#include<iostream>
#include<fstream>

__global__ void _run_neurongroup_group_variable_set_conditional_codeobject_1_kernel(
	unsigned int _neurongroup_N,
	unsigned int max_threads_per_block,
	float* _array_rands,
	double* _array_neurongroup_w,
	bool* _array_neurongroup_not_refractory)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;

	int neuron_id = bid*max_threads_per_block + tid;
	if(neuron_id < 0 || neuron_id >= _neurongroup_N)
	{
		return;
	}

	bool _cond = true;
	if(_cond)
	{
		const bool not_refractory = _array_neurongroup_not_refractory[neuron_id];
		double w = _array_neurongroup_w[neuron_id];
		if(not_refractory)
		{
			float r = _array_rands[neuron_id];	//get random pregenerated number
			w = r * 10 * 0.0001;
			_array_neurongroup_w[neuron_id] = w;
		}
	}
}

void _run_neurongroup_group_variable_set_conditional_codeobject_1()
{
	using namespace brian;

	//genenerate an arry of random numbers on the device
	float* dev_array_rands;
	hipMalloc((void**)&dev_array_rands, sizeof(float)*neurongroup_N);
	if(!dev_array_rands)
	{
		printf("ERROR while allocating device memory with size %ld in _run_neurongroup_group_variable_set_conditional_codeobject_1()\n", sizeof(float)*neurongroup_N);
	}
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, time(0));
	hiprandGenerateUniform(gen, dev_array_rands, neurongroup_N);

	unsigned int blocks = (neurongroup_N + max_threads_per_block - 1)/max_threads_per_block;	// = ceil(N/num_threads)

	_run_neurongroup_group_variable_set_conditional_codeobject_1_kernel<<<blocks, max_threads_per_block>>>(
		neurongroup_N,
		max_threads_per_block,
		dev_array_rands,
		dev_array_neurongroup_w,
		dev_array_neurongroup_not_refractory);

	hiprandDestroyGenerator(gen);
	hipFree(dev_array_rands);
}

