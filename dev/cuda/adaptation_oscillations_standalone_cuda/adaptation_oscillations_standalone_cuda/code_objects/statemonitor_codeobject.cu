#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/statemonitor_codeobject.h"
#include<math.h>
#include "brianlib/common_math.h"
#include<stdint.h>
#include<iostream>
#include<fstream>


__global__ void _run_statemonitor_codeobject_kernel(
	int _neuron_id,
	int _index_last_element,
	double* _statemonitor_v,
	double* _statemonitor_w,
	double* _array_neurongroup_v,
	double* _array_neurongroup_w
	)
{
	_statemonitor_v[_index_last_element] = _array_neurongroup_v[_neuron_id];
	_statemonitor_w[_index_last_element] = _array_neurongroup_w[_neuron_id];
}

void _run_statemonitor_codeobject()
{
	using namespace brian;

	double t = defaultclock.t_();
	int num_indices = _num__array_statemonitor__indices;
	_dynamic_array_statemonitor_t.push_back(t);

	for(int i = 0; i < num_indices; i++)
	{
		unsigned int neuron_id = _static_array__array_statemonitor__indices[i];
		_dynamic_array_statemonitor__recorded_v[i].push_back(0.0);	//push dummy value
		_dynamic_array_statemonitor__recorded_w[i].push_back(0.0);	//push dummy value
		double* dev_dynamic_array_statemonitor__recorded_v = thrust::raw_pointer_cast(&(_dynamic_array_statemonitor__recorded_v[i][0]));
		double* dev_dynamic_array_statemonitor__recorded_w = thrust::raw_pointer_cast(&(_dynamic_array_statemonitor__recorded_w[i][0]));
		int index_last_element = _dynamic_array_statemonitor__recorded_v[i].size() - 1;
		_run_statemonitor_codeobject_kernel<<<1, 1>>>(
			neuron_id,
			index_last_element,
			dev_dynamic_array_statemonitor__recorded_v,
			dev_dynamic_array_statemonitor__recorded_w,
			dev_array_neurongroup_v,
			dev_array_neurongroup_w);
	}
}

