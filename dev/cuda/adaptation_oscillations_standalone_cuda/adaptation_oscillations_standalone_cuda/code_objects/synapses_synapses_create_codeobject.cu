#include "objects.h"
#include "code_objects/synapses_synapses_create_codeobject.h"
#include<math.h>
#include "brianlib/common_math.h"
#include<stdint.h>
#include<iostream>
#include<fstream>

#include <inttypes.h>
#include "brianlib/synapses.h"

#define N 4000
#define N_squared (N*N)

void _run_synapses_synapses_create_codeobject()
{
	using namespace brian;

	float* _array_random;
	_array_random = (float*)malloc(sizeof(float)*N_squared);
	hiprandGenerator_t gen;
	hiprandCreateGeneratorHost(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, time(0));
	hiprandGenerateUniform(gen, _array_random, N_squared);

	///// CONSTANTS ///////////
	const int _num_all_post = N;
	const int _num_synaptic_post = _dynamic_array_synapses__synaptic_post.size();
	const int _num_all_pre = N;
	const int _num_synaptic_pre = _dynamic_array_synapses__synaptic_pre.size();
	///// POINTERS ////////////
	int32_t * __restrict__ _ptr_array_synapses_N_incoming = _array_synapses_N_incoming;
	int32_t * __restrict__ _ptr_array_synapses_N_outgoing = _array_synapses_N_outgoing;
	int32_t * __restrict__ _ptr_array_neurongroup_i = _array_neurongroup_i;

	std::vector<int32_t> temp_pre;
	std::vector<int32_t> temp_post;

	int _synapse_idx = _dynamic_array_synapses__synaptic_pre.size();
	for(int i=0; i<_num_all_pre; i++)
	{
		for(int j=0; j<_num_all_post; j++)
		{
			const int32_t _all_post = _ptr_array_neurongroup_i[j];
			const int32_t _all_pre = _ptr_array_neurongroup_i[i];
			const int32_t _pre_idx = _all_pre;
			const int32_t _post_idx = _all_post;
			const bool _cond = i != j;
			const int32_t _n = 1;
//			const double _p = 0.05;
			const double _p = 0.005;
			// Add to buffer
			if(_cond)
			{
				if (_p != 1.0)
				{
					// We have to use _rand instead of rand to use our rand
					// function, not the one from the C standard library
					if (_array_random[i*_num_all_post + j] >= _p)
					    continue;
				}

				for (int _repetition=0; _repetition<_n; _repetition++)
				{
					_ptr_array_synapses_N_outgoing[_pre_idx] += 1;
					_ptr_array_synapses_N_incoming[_post_idx] += 1;
					temp_pre.push_back(_pre_idx);
					temp_post.push_back(_post_idx);
					_synapse_idx++;
				}
			}
		}
	}

	_dynamic_array_synapses__synaptic_post = temp_post;
	_dynamic_array_synapses__synaptic_pre = temp_pre;

	// now we need to resize all registered variables
	const int newsize = _dynamic_array_synapses__synaptic_pre.size();
	_dynamic_array_synapses__synaptic_post.resize(newsize);
	_dynamic_array_synapses__synaptic_pre.resize(newsize);
	_dynamic_array_synapses_c.resize(newsize);
	_dynamic_array_synapses_pre_delay.resize(newsize);
	_dynamic_array_synapses_lastupdate.resize(newsize);
	// Also update the total number of synapses
	synapses._N_value = newsize;
}


