#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/neurongroup_thresholder_codeobject.h"
#include<math.h>
#include "brianlib/common_math.h"
#include<stdint.h>
#include<iostream>
#include<fstream>

#define N 4000
#define ceil(N, num) ((N + num-1)/num)

__global__ void _run_neurongroup_thresholder_codeobject_kernel(int stride, double par_t, int32_t* par_array_neurongroup__spikespace, double* par_array_neurongroup_v, double* par_array_neurongroup_lastspike, bool* par_array_neurongroup_not_refractory)
{
	using namespace brian;

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	if(bid*stride + tid >= N)
		return;

	double t = par_t;
	int32_t* array_neurongroup__spikespace = par_array_neurongroup__spikespace;
	double* array_neurongroup_v = par_array_neurongroup_v;
	double* array_neurongroup_lastspike = par_array_neurongroup_lastspike;
	bool* array_neurongroup_not_refractory = par_array_neurongroup_not_refractory;


	array_neurongroup__spikespace[bid * stride + tid] = -1;
	if(tid == bid && bid == 0)
	{
		array_neurongroup__spikespace[N] = 0;
	}

	if(tid == 0)
	{
		int num_spikes = 0;

		for(int i = bid * stride; i < (bid + 1)*stride; i++)
		{
			const double v = array_neurongroup_v[i];
			const bool not_refractory = array_neurongroup_not_refractory[i];
			const double _cond = (v > 0.001) && (not_refractory);
			if(_cond && i < N)
			{
				array_neurongroup__spikespace[bid*stride + num_spikes] = i;
				array_neurongroup_not_refractory[i] = false;
				array_neurongroup_lastspike[i] = t;
				num_spikes++;
			}
		}
		atomicAdd(&array_neurongroup__spikespace[N], num_spikes);
	}
}

void _run_neurongroup_thresholder_codeobject()
{
	using namespace brian;

	const double t = defaultclock.t_();

	//// MAIN CODE ////////////
	_run_neurongroup_thresholder_codeobject_kernel<<<num_blocks_sequential, ceil(N, num_blocks_sequential)>>>(ceil(N, num_blocks_sequential), t, dev_array_neurongroup__spikespace, dev_array_neurongroup_v, dev_array_neurongroup_lastspike, dev_array_neurongroup_not_refractory);
}


