#include "objects.h"
#include "random.h"

#include <hiprand.h>

#define neuron_N 4000

void _random_number_generation()
{
	using namespace brian;

	float mean = 0.0;
	float std_deviation = 1.0;
	hiprandGenerateNormal(random_float_generator, dev_array_random_floats, neuron_N, mean, std_deviation);
}
