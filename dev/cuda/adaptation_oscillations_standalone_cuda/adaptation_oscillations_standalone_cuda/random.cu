#include "objects.h"
#include "random.h"
#include <hiprand.h>

#define N 4000

void _get_random()
{
	using namespace brian;
	hiprandGenerateNormal(gen, dev_array_random, N, 0.0, 1.0);
}
