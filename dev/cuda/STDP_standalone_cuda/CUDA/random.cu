#include "objects.h"
#include "random.h"
#include <hiprand.h>

#define N 1000

void _get_rands()
{
	using namespace brian;
	hiprandGenerateUniform(gen, dev_array_rands, N);
}
