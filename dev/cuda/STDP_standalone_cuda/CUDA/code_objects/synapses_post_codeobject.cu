#include "hip/hip_runtime.h"
#include "objects.h"
#include "code_objects/synapses_post_codeobject.h"
#include<math.h>
#include "brianlib/common_math.h"
#include<stdint.h>
#include<iostream>
#include<fstream>

#include <inttypes.h>
#include "brianlib/synapses.h"

#include <thrust/device_vector.h>

////// SUPPORT CODE ///////
namespace {
	__device__ double _clip(const float value, const float a_min, const float a_max)
	{
	    if (value < a_min)
	        return a_min;
	    if (value > a_max)
	        return a_max;
	    return value;
	}
}

////// HASH DEFINES ///////


__global__ void _run_synapses_post_pre_codeobject_kernel(int par_num_threads, int par_num_syn)
{
	int tid = threadIdx.x;

	CudaVector<int32_t>* synapses_queue;
	CudaVector<int32_t>* pre_neuron_queue;
	CudaVector<int32_t>* post_neuron_queue;

	int num_threads = par_num_threads;
	int num_syn = par_num_syn;
	int lower = tid*(num_syn/num_threads);
	int upper = (tid + 1)*(num_syn/num_threads);

	brian::synapses_post.queue->peek(&synapses_queue, &pre_neuron_queue, &post_neuron_queue);

	for(int i = 0; i < brian::synapses_post.queue->num_parallel; i++)
	{
		for(int j = 0; j < pre_neuron_queue[i].size(); j++)
		{
			int32_t pre_idx = pre_neuron_queue[i].get(j);
			if(pre_idx >= lower && pre_idx < upper)
			{
				//DO NOTHING
			}
		}
	}
}

__global__ void _run_synapses_post_syn_codeobject_kernel(int par_num_threads, int par_num_syn, double* par_array_synapses_Apre, int par_numApre,
	double* par_array_synapses_lastupdate, int par_numlastupdate,
	double* par_array_synapses_Apost, int par_numApost, double* par_array_synapses_w,
	int par_numw, int32_t* par_array_synapses__synaptic_pre, int par_num_synaptic_pre,
	double par_t)
{
	int tid = threadIdx.x;

	CudaVector<int32_t>* synapses_queue;
	CudaVector<int32_t>* pre_neuron_queue;
	CudaVector<int32_t>* post_neuron_queue;

	int num_threads = par_num_threads;
	int num_syn = par_num_syn;
	int lower = tid*(num_syn/num_threads);
	int upper = (tid + 1)*(num_syn/num_threads);

	brian::synapses_post.queue->peek(&synapses_queue, &pre_neuron_queue, &post_neuron_queue);

	double * _ptr_array_synapses_Apre = par_array_synapses_Apre;
	//int _numApre = par_numApre;
	double * _ptr_array_synapses_lastupdate = par_array_synapses_lastupdate;
	//int _numlastupdate = par_numlastupdate;
	double * _ptr_array_synapses_Apost = par_array_synapses_Apost;
	//int _numApost = par_numApost;
	double * _ptr_array_synapses_w = par_array_synapses_w;
	//int _numw = par_numw;
	//int32_t * _ptr_array_synapses__synaptic_pre = par_array_synapses__synaptic_pre;
	//int _num_synaptic_pre = par_num_synaptic_pre;
	double t = par_t;

	//iterate over all queues
	for(int i = 0; i < brian::synapses_post.queue->num_parallel; i++)
	{
		//and over all elements in each queue
		for(int j = 0; j < synapses_queue[i].size(); j++)
		{
			int32_t syn_idx = synapses_queue[i].get(j);
			//we are only responsible for parts of the work
			if(syn_idx >= lower && syn_idx < upper)
			{
				const int32_t syn_idx = synapses_queue[i].get(j);
				double Apre = _ptr_array_synapses_Apre[syn_idx];
				double lastupdate = _ptr_array_synapses_lastupdate[syn_idx];
				double Apost = _ptr_array_synapses_Apost[syn_idx];
				double w = _ptr_array_synapses_w[syn_idx];
				Apre = Apre * exp(-(t - lastupdate) / 0.02);
				Apost = Apost * exp(-(t - lastupdate) / 0.02);
				Apost += -0.000105;
				w = _clip(w + Apre, 0, 0.01);
				lastupdate = t;
				_ptr_array_synapses_Apre[syn_idx] = Apre;
				_ptr_array_synapses_lastupdate[syn_idx] = lastupdate;
				_ptr_array_synapses_Apost[syn_idx] = Apost;
				_ptr_array_synapses_w[syn_idx] = w;
			}
		}
	}
}

__global__ void _run_synapses_post_post_codeobject_kernel(int par_num_threads, int par_num_syn)
{
	int tid = threadIdx.x;

	CudaVector<int32_t>* synapses_queue;
	CudaVector<int32_t>* pre_neuron_queue;
	CudaVector<int32_t>* post_neuron_queue;

	int num_threads = par_num_threads;
	int num_syn = par_num_syn;
	int lower = tid*(num_syn/num_threads);
	int upper = (tid + 1)*(num_syn/num_threads);

	brian::synapses_post.queue->peek(&synapses_queue, &pre_neuron_queue, &post_neuron_queue);

	for(int i = 0; i < brian::synapses_post.queue->num_parallel; i++)
	{
		for(int j = 0; j < post_neuron_queue[i].size(); j++)
		{
			int32_t post_idx = post_neuron_queue[i].get(j);
			if(post_idx >= lower && post_idx < upper)
			{
				//DO NOTHING
			}
		}
	}
}

void _run_synapses_post_codeobject()
{
	using namespace brian;
	///// CONSTANTS ///////////
	const int _numApre = _dynamic_array_synapses_Apre.size();
	const int _numlastupdate = _dynamic_array_synapses_lastupdate.size();
	const int _numApost = _dynamic_array_synapses_Apost.size();
	const int _numw = _dynamic_array_synapses_w.size();
	const int _num_synaptic_pre = _dynamic_array_synapses__synaptic_pre.size();
	const double t = defaultclock.t_();

	int max_num_threads = num_blocks_sequential;

	// This is only needed for the _debugmsg function below
	double* dev_array_synapses_Apre = thrust::raw_pointer_cast(&_dynamic_array_synapses_Apre[0]);
	double* dev_array_synapses_lastupdate = thrust::raw_pointer_cast(&_dynamic_array_synapses_lastupdate[0]);
	double* dev_array_synapses_Apost = thrust::raw_pointer_cast(&_dynamic_array_synapses_Apost[0]);
	double* dev_array_synapses_w = thrust::raw_pointer_cast(&_dynamic_array_synapses_w[0]);
	int32_t* dev_array_synapses__synaptic_pre = thrust::raw_pointer_cast(&_dynamic_array_synapses__synaptic_pre[0]);

	_run_synapses_post_pre_codeobject_kernel<<<max_num_threads, 1>>>(max_num_threads, 1000);

	_run_synapses_post_syn_codeobject_kernel<<<max_num_threads, 1>>>(max_num_threads, 1, dev_array_synapses_Apre,
		_numApre, dev_array_synapses_lastupdate, _numlastupdate, dev_array_synapses_Apost,
		_numApost, dev_array_synapses_w, _numw, dev_array_synapses__synaptic_pre,
		_num_synaptic_pre, t);

	_run_synapses_post_post_codeobject_kernel<<<max_num_threads,1>>>(max_num_threads, 1);
}

void _debugmsg_synapses_post_codeobject()
{
	using namespace brian;
	std::cout << "Number of synapses: " << _dynamic_array_synapses__synaptic_pre.size() << endl;
}

