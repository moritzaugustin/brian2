#include "hip/hip_runtime.h"
#include "objects.h"


#include "code_objects/synapses_post_push_spikes.h"
#include <math.h>
#include <stdint.h>
#include "brianlib/common_math.h"

__global__ void _run_synapses_post_push_spikes_advance_kernel()
{
	int tid = threadIdx.x;
	brian::synapses_post.queue->advance(tid);
}

__global__ void _run_synapses_post_push_spikes_push_kernel(int* spikespace)
{
	int tid = threadIdx.x;
	brian::synapses_post.queue->push(tid, spikespace, spikespace[1]);
}

void _run_synapses_post_push_spikes()
{
	using namespace brian;
	///// CONSTANTS ///////////
	//const int _num_spikespace = 2;
	///// POINTERS ////////////

	int32_t * __restrict__ _ptr_array_neurongroup__spikespace = _array_neurongroup__spikespace;

	//// MAIN CODE ////////////
	// we do advance at the beginning rather than at the end because it saves us making
	// a copy of the current spiking synapses
	_run_synapses_post_push_spikes_advance_kernel<<<1, num_multiprocessors>>>();
	_run_synapses_post_push_spikes_push_kernel<<<1, num_multiprocessors>>>(_ptr_array_neurongroup__spikespace);
}
